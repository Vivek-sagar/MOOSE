#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "GpuLookup.h"


GpuLookupTable::GpuLookupTable()
{

}

GpuLookupTable::GpuLookupTable(double *min, double *max, int *nDivs, unsigned int nSpecies)
{
	// min_ = *min;
	// max_ = *max;
	// nPts_ = *nDivs + 1 + 1;
	// dx_= ( *max - *min ) / *nDivs;
	// nColumns_ = 2 * nSpecies;

	hipMalloc((void **)&min_d, sizeof(double));
	hipMalloc((void **)&max_d, sizeof(double));
	hipMalloc((void **)&nPts_d, sizeof(unsigned int));
	hipMalloc((void **)&dx_d, sizeof(double));
	hipMalloc((void **)&nColumns_d, sizeof(unsigned int));

	// Number of points is 1 more than number of divisions.
	// Then add one more since we may interpolate at the last point in the table.
	// Every row has 2 entries for each type of gate

	unsigned int nPts_ = *nDivs + 1 + 1;
	double dx_= ( *max - *min ) / *nDivs;
	unsigned int nColumns_ = 2 * nSpecies;

 	hipMemcpy( min_d, min, sizeof(double), hipMemcpyHostToDevice);
 	hipMemcpy( max_d, max, sizeof(double), hipMemcpyHostToDevice);
 	hipMemcpy( nPts_d, &nPts_, sizeof(unsigned int), hipMemcpyHostToDevice);
 	hipMemcpy( dx_d, &dx_, sizeof(double), hipMemcpyHostToDevice);
 	hipMemcpy( nColumns_d, &nColumns_, sizeof(unsigned int), hipMemcpyHostToDevice);

 	// Just randomly assumes that there will be only 50 species and allocates memory for that. BAD IDEA!
	hipMalloc((void **)&table_d, (nPts_ * 100) * sizeof(double));

}

void GpuLookupTable::sayHi()
{
	std::cout << "Hi there! ";
}

void GpuLookupTable::addColumns(int species, double *C1, double *C2)
{
	// double *iTable = table_d+nColumns_;
	// for (int i=0)

}

void GpuLookupTable::lookup(double row, double column, double& C1, double& C2)
{

}
