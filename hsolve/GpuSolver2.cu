#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "GpuSolver.h"

using namespace std;
 
__global__ 

void findSumToN(int *n, int limit)
{
	int tId = threadIdx.x;
	
	for (int i=0; i<=(int)log2((double)limit); i++)
	{
		if (tId%(int)(pow(2.0,(double)(i+1))) == 0){
			if (tId+(int)pow(2.0, (double)i) >= limit) break;
			n[tId] += n[tId+(int)pow(2.0, (double)i)];
		}
		__syncthreads();
	}
}

GpuInterface::GpuInterface()
{
	y = 20;
	asize = y*sizeof(int);
	for (int i=0; i<y; i++)
		n[i] = i;
}

void GpuInterface::sayHi()
{
	cout << "Hello there\n";
}

void GpuInterface::lookupTables(double &v, double* A, double* B) const
{
	if (v <= xmin_){
		*A = A[0];
		*B = B[0];
	}
	else if (v >= xmax_){
		*A = A[ASize_];
		*B = B[BSize_];
	}
	else{	
		unsigned int index = (v-xmin_) * invDx_;
		//assert(ASize_ > index && BSize_ > index);
		//Check for lookupByInterpolation in the HHGate code
		double frac = (v-xmin_-(index/invDx_)) * invDx_;
		*A = A_[index]*(1-frac) + A_[index+1] * frac;
		*B = B_[index]*(1-frac) + B_[index+1] * frac;
	}
}

void GpuInterface::setupTables(double *A, double *B, double ASize, double BSize, double xmin, double xmax, double invDx)
{
	hipMalloc( (void**)&A_d, ASize*sizeof(double));
	hipMalloc( (void**)&B_d, BSize*sizeof(double));
	hipMalloc( (void**)&xmin_d, sizeof(double));
	hipMalloc( (void**)&xmax_d, sizeof(double));
	hipMalloc( (void**)&invDx_d, sizeof(double));
	hipMalloc( (void**)&ASize_d, sizeof(int));
	hipMalloc( (void**)&BSize_d, sizeof(int));

	hipMemcpy(A_d, A, ASize*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, BSize*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(xmin_d, &xmin, BSize*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(xmax_d, xmax, BSize*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, BSize*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, BSize*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, BSize*sizeof(double), hipMemcpyHostToDevice);
}


int GpuInterface::calculateSum()
{
	int *n_d;
	hipMalloc( (void**)&n_d, asize );

        hipMemcpy(n_d, n, asize, hipMemcpyHostToDevice );

        dim3 dimBlock( y, 1 );
        dim3 dimGrid( 1, 1 );
        findSumToN<<<dimGrid, dimBlock>>>(n_d, y);
        hipMemcpy(n, n_d, asize, hipMemcpyDeviceToHost);
        hipFree (n_d);
        return n[0];
}

void GpuInterface::setY(int newVal)
{
	y = newVal;
	asize = y*sizeof(int);
	for (int i=0; i<y; i++)
                n[i] = i;

}
/*
int main()
{
        GpuInterface obj;
        obj.setY(20);
        std::cout << obj.calculateSum();
        return EXIT_SUCCESS;
}
*/
