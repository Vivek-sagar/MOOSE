#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "GpuSolver.h"
 
__global__ 

void findSumToN(int *n, int limit)
{
	int tId = threadIdx.x;
	
	for (int i=0; i<=(int)log2((double)limit); i++)
	{
		if (tId%(int)(pow(2.0,(double)(i+1))) == 0){
			if (tId+(int)pow(2.0, (double)i) >= limit) break;
			n[tId] += n[tId+(int)pow(2.0, (double)i)];
		}
		__syncthreads();
	}
}

GpuInterface::GpuInterface()
{
	y = 20;
	asize = y*sizeof(int);
	for (int i=0; i<y; i++)
		n[i] = i;
}

int GpuInterface::calculateSum()
{
	int *n_d;
	hipMalloc( (void**)&n_d, asize );

        hipMemcpy(n_d, n, asize, hipMemcpyHostToDevice );

        dim3 dimBlock( y, 1 );
        dim3 dimGrid( 1, 1 );
        findSumToN<<<dimGrid, dimBlock>>>(n_d, y);
        hipMemcpy(n, n_d, asize, hipMemcpyDeviceToHost);
        hipFree (n_d);
        return n[0];
}

void GpuInterface::setY(int newVal)
{
	y = newVal;
	asize = y*sizeof(int);
	for (int i=0; i<y; i++)
                n[i] = i;

}
/*
int main()
{
        GpuInterface obj;
        obj.setY(20);
        std::cout << obj.calculateSum();
        return EXIT_SUCCESS;
}
*/
